
#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void loop()
{
   int i = blockIdx.x * blockDim.x + threadIdx.x;
   printf("%d\n", i);
}

int main()
{
  loop<<<5,2>>>();
  hipDeviceSynchronize();
}
