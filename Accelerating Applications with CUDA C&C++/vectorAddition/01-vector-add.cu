
#include <hip/hip_runtime.h>
#include <stdio.h>

void initWith(float num, float *a, int N)
{
  for(int i = 0; i < N; ++i)
  {
    a[i] = num;
  }
}

__global__ void addVectorsInto(float *result, float *a, float *b, int N)
{


    int inx = threadIdx.x + blockIdx.x * blockDim.x;
    int gridStride = gridDim.x * blockDim.x;

  for(int i = inx; i < N; i += gridStride)
  {
    result[i] = a[i] + b[i];
  }
}

void checkElementsAre(float target, float *array, int N)
{
  for(int i = 0; i < N; i++)
  {
    if(array[i] != target)
    {
      printf("FAIL: array[%d] - %0.0f does not equal %0.0f\n", i, array[i], target);
      exit(1);
    }
  }
  printf("SUCCESS! All values added correctly.\n");
}

int main()
{
  const int N = 2<<20;
  size_t size = N * sizeof(float);


  size_t threads_per_block = 512;

  size_t number_of_blocks = (N + threads_per_block - 1) / threads_per_block;

 

  float *a;
  float *b;
  float *c;

  hipMallocManaged(&a, size);
  hipMallocManaged(&b, size);
  hipMallocManaged(&c, size);

  initWith(3, a, N);
  initWith(4, b, N);
  initWith(0, c, N);

  addVectorsInto<<<number_of_blocks, threads_per_block>>>(c, a, b, N);
  hipDeviceSynchronize();

  checkElementsAre(7, c, N);

  hipFree(a);
  hipFree(b);
  hipFree(c);
}
